#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <chrono>
#include <string>
// ============================================================================
// main function
//
// Modifications
//    Chien  21-12-29    initial developments
//
// ============================================================================
// -- nvidia cuda --
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// -- magma --
#include "magma_v2.h"

// -- input cmd --
//#include "magmaHC/input-info.h"

// -- magma --
#include "magmaHC/magmaHC-problems.cuh"

// -- p2c --
#include "magmaHC/const-matrices/p2c-3view_unknownf_pHC.h"
#include "magmaHC/const-matrices/p2c-4vTrg.h"
#include "magmaHC/const-matrices/p2c-3vTrg.h"
#include "magmaHC/const-matrices/p2c-3vTrg_relax.h"
#include "magmaHC/const-matrices/p2c-5pt_rel_pose_w_depth_recon.h"
#include "magmaHC/const-matrices/p2c-optimalPnP_w_quaternion.h"
#include "magmaHC/const-matrices/p2c-3pt_rel_pose_w_homo_constraint.h"
#include "magmaHC/const-matrices/p2c-r6p.h"
#include "magmaHC/const-matrices/p2c-refractive_p5p.h"
#include "magmaHC/const-matrices/p2c-refractive_p6p.h"

// -- global repo directory --
std::string repo_dir = "/users/cchien3/data/cchien3/MyBitBucket/issac-parametric-hc/";

int main(int argc, char **argv) {
  --argc; ++argv;
  std::string arg;
  int argIndx = 0;
  int argTotal = 4;
  std::string HC_problem = "default";

  if (argc) {
    arg = std::string(*argv);
    if (arg == "-h" || arg == "--help") {
      magmaHCWrapper::print_usage();
      exit(1);
    }
    else if (argc <= argTotal) {
      while(argIndx <= argTotal-1) {
        if (arg == "-p" || arg == "--problem") {
          argv++;
          arg = std::string(*argv);
          HC_problem = arg;
          argIndx+=2;
          break;
        }
        else {
          std::cerr<<"invalid input arguments! See examples: \n";
          magmaHCWrapper::print_usage();
          exit(1);
        }
        argv++;
      }
    }
    else if (argc > argTotal) {
      std::cerr<<"too many arguments!\n";
      magmaHCWrapper::print_usage();
      exit(1);
    }
  }
  else {
    magmaHCWrapper::print_usage();
    exit(1);
  }

  magmaFloatComplex *h_startSols;
  magmaFloatComplex *h_Track;
  magmaFloatComplex *h_startParams;
  magmaFloatComplex *h_targetParams;
  magmaFloatComplex *h_phc_coeffs_Hx;
  magmaFloatComplex *h_phc_coeffs_Ht;
  magma_int_t *h_Hx_idx;
  magma_int_t *h_Ht_idx;

  // -- files to be read --
  std::string repo_root_dir = repo_dir;
  repo_dir.append("problems/");
  std::string problem_filename = repo_dir.append(HC_problem);

  // -- declare class objects (put the long lasting object in dynamic memory) --
  magmaHCWrapper::problem_params* pp = new magmaHCWrapper::problem_params;
  magmaHCWrapper::const_mats* cm = new magmaHCWrapper::const_mats;

  pp->define_problem_params(problem_filename, HC_problem);

  // -- allocate tracks and coeffs arrays in cpu --
  magma_cmalloc_cpu( &h_startSols, pp->numOfTracks*(pp->numOfVars+1) );
  magma_cmalloc_cpu( &h_Track, pp->numOfTracks*(pp->numOfVars+1) );
  magma_cmalloc_cpu( &h_startParams, pp->numOfParams );
  magma_cmalloc_cpu( &h_targetParams, pp->numOfParams );

  magma_cmalloc_cpu( &h_phc_coeffs_Hx, pp->numOfCoeffsFromParams*(pp->max_orderOf_t+1) );
  magma_cmalloc_cpu( &h_phc_coeffs_Ht, pp->numOfCoeffsFromParams*(pp->max_orderOf_t) );
  magma_imalloc_cpu( &h_Hx_idx, pp->numOfVars*pp->numOfVars*pp->Hx_maximal_terms*pp->Hx_maximal_parts );
  magma_imalloc_cpu( &h_Ht_idx, pp->numOfVars*pp->Ht_maximal_terms*pp->Ht_maximal_parts );

  // =============================================================================
  // -- read files: start solutions, start coefficients, and target parameters --
  // =============================================================================
  std::string targetParam_filename_test = problem_filename;
  std::string startParams_filename_test = problem_filename;
  std::string startSols_filename_test = problem_filename;
  startSols_filename_test.append("/start_sols.txt");
  targetParam_filename_test.append("/target_params.txt");
  startParams_filename_test.append("/start_params.txt");
  std::fstream startCoef_file;
  std::fstream targetParams_file;
  std::fstream startSols_file;
  bool read_success = 0;
  bool start_sols_read_success = 0;
  bool start_coeffs_read_success = 0;
  bool targetParams_read_success = 0;
  
  float s_real, s_imag;
  int d = 0, i = 0; 
  startSols_file.open(startSols_filename_test, std::ios_base::in);
  std::cout<<"============= start solutions h_startSols ============"<<std::endl;
  if (!startSols_file) {
    std::cerr << "problem start solutions file not existed!\n";
  }
  else {
    while (startSols_file >> s_real >> s_imag) {
      (h_startSols + i * (pp->numOfVars+1))[d] = MAGMA_C_MAKE(s_real, s_imag);
      (h_Track + i * (pp->numOfVars+1))[d] = MAGMA_C_MAKE(s_real, s_imag);
      if (d < pp->numOfVars-1) {
        d++;
      }
      else {
        d = 0;
        i++;
      }
    }
    for(int k = 0; k < pp->numOfTracks; k++) {
      (h_startSols + k * (pp->numOfVars+1))[pp->numOfVars] = MAGMA_C_MAKE(1.0, 0.0);
      (h_Track + k * (pp->numOfVars+1))[pp->numOfVars] = MAGMA_C_MAKE(1.0, 0.0);
    }
    start_sols_read_success = 1;
  }

  // -- read start system coefficients --
  d = 0;
  startCoef_file.open(startParams_filename_test, std::ios_base::in);
  if (!startCoef_file) {
    std::cerr << "problem start coefficients file not existed!\n";
  }
  else {
    while (startCoef_file >> s_real >> s_imag) {
      (h_startParams)[d] = MAGMA_C_MAKE(s_real, s_imag);
      d++;
    }
    start_coeffs_read_success = 1;
  }

  d = 0;
  targetParams_file.open(targetParam_filename_test, std::ios_base::in);
  if (!targetParams_file) {
    std::cerr << "problem target parameters file not existed!\n";
  }
  else {
    while (targetParams_file >> s_real >> s_imag) {
      (h_targetParams)[d] = MAGMA_C_MAKE(s_real, s_imag);
      d++;
    }
    targetParams_read_success = 1;
  }

  // ---------------------------------------------------------------------------------------------------
  bool Hx_file_read_success = false;
  bool Ht_file_read_success = false;
  
  std::string filename_Hx = problem_filename;
  std::string filename_Ht = problem_filename;
  filename_Hx.append("/Hx_idx.txt");
  filename_Ht.append("/Ht_idx.txt");
  std::fstream Hx_idx_file;
  std::fstream Ht_idx_file;
  
  // -- 4) read Hx index matrix, if required --
  int index;
  d = 0;
  Hx_idx_file.open(filename_Hx, std::ios_base::in);
  if (!Hx_idx_file) {
    std::cerr << "problem Hx index matrix file not existed!\n";
  }
  else {
    while (Hx_idx_file >> index) {
      (h_Hx_idx)[d] = index;
      d++;
    }
    Hx_file_read_success = 1;
  }
  // -- 5) read Ht index matrix --
  d = 0;
  Ht_idx_file.open(filename_Ht, std::ios_base::in);
  if (!Ht_idx_file) {
    std::cerr << "problem Ht index matrix file not existed!\n";
  }
  else {
    while (Ht_idx_file >> index) {
      (h_Ht_idx)[d] = index;
      d++;
    }
    Ht_file_read_success = 1;
  }

  magma_cprint(pp->numOfParams, 1, h_targetParams, pp->numOfParams);

  // -- params2coeffs --
  // -- coefficients of function t for H, used in gpu --
  if(HC_problem == "3view_unknownf_pHC")
    magmaHCWrapper::p2c_3view_unknwonf_pHC(h_targetParams, h_startParams, h_phc_coeffs_Hx, h_phc_coeffs_Ht);
  else if (HC_problem == "4vTrg")
    magmaHCWrapper::p2c_4vTrg(h_targetParams, h_startParams, h_phc_coeffs_Hx, h_phc_coeffs_Ht);
  else if (HC_problem == "3vTrg")
    magmaHCWrapper::p2c_3vTrg(h_targetParams, h_startParams, h_phc_coeffs_Hx, h_phc_coeffs_Ht);
  else if (HC_problem == "3vTrg_relax")
    magmaHCWrapper::p2c_3vTrg_relax(h_targetParams, h_startParams, h_phc_coeffs_Hx, h_phc_coeffs_Ht);
  else if (HC_problem == "5pt_rel_pose_w_depth_recon")
    magmaHCWrapper::p2c_5pt_rel_pose_w_depth_recon(h_targetParams, h_startParams, h_phc_coeffs_Hx, h_phc_coeffs_Ht);
  else if (HC_problem == "optimalPnP_w_quaternion")
    magmaHCWrapper::p2c_optimalPnP_w_quaternion(h_targetParams, h_startParams, h_phc_coeffs_Hx, h_phc_coeffs_Ht);
  else if (HC_problem == "3pt_rel_pose_w_homo_constraint")
    magmaHCWrapper::p2c_3pt_rel_pose_w_homo_constraint(h_targetParams, h_startParams, h_phc_coeffs_Hx, h_phc_coeffs_Ht);
  else if (HC_problem == "r6p")
    magmaHCWrapper::p2c_r6p(h_targetParams, h_startParams, h_phc_coeffs_Hx, h_phc_coeffs_Ht);
  else if (HC_problem == "refractive_p5p")
    magmaHCWrapper::p2c_refractive_p5p(h_targetParams, h_startParams, h_phc_coeffs_Hx, h_phc_coeffs_Ht);
  else if (HC_problem == "refractive_p6p")
    magmaHCWrapper::p2c_refractive_p6p(h_targetParams, h_startParams, h_phc_coeffs_Hx, h_phc_coeffs_Ht);

  read_success = (start_sols_read_success && start_coeffs_read_success && targetParams_read_success && Hx_file_read_success && Ht_file_read_success);

  // -- write successful track solutions to files --
  std::ofstream track_sols_file;
  std::string write_sols_file_dir = repo_root_dir;
  write_sols_file_dir.append("converged_HC_tracks.txt");
  track_sols_file.open(write_sols_file_dir);
  if ( !track_sols_file.is_open() )
    std::cout<<"successful solutions for evaluation write files cannot be opened!"<<std::endl;

  // -- call homotopy continuation solver --
  //read_success = (start_sols_read_success && start_coeffs_read_success && targetParams_read_success);
  if (read_success) {
    magmaHCWrapper::homotopy_continuation_solver(h_startSols, h_Track, h_startParams, h_targetParams, h_Hx_idx, h_Ht_idx, h_phc_coeffs_Hx, h_phc_coeffs_Ht, pp, cm, HC_problem, track_sols_file);
  }
  else {
    std::cout<<"read files failed!"<<std::endl;
    exit(1);
  }

  delete pp;
  delete cm;
  magma_free_cpu( h_startSols );
  magma_free_cpu( h_Track );
  magma_free_cpu( h_startParams );
  magma_free_cpu( h_targetParams );
  magma_free_cpu( h_phc_coeffs_Hx );
  magma_free_cpu( h_phc_coeffs_Ht );

  magma_free_cpu( h_Hx_idx );
  magma_free_cpu( h_Ht_idx );

  track_sols_file.close();

  return 0;
}
