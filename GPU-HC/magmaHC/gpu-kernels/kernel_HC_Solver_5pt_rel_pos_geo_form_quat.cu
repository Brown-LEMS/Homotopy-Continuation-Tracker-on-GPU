#include "hip/hip_runtime.h"
#ifndef kernel_HC_Solver_5pt_rel_pos_geo_form_quat_cu
#define kernel_HC_Solver_5pt_rel_pos_geo_form_quat_cu
// =======================================================================================
// GPU homotopy continuation solver for 5-point relative pose problem (Geometric Form)
//
// Modifications
//    Chiang-Heng Chien  22-10-31:   Initially created
//    Chiang-Heng Chien  24-01-04:   Add macro definitions for computing coefficients from parameter homotopy
//
//> (c) LEMS, Brown University
//> Chiang-Heng Chien (chiang-heng_chien@brown.edu)
// =======================================================================================
#include <stdio.h>
#include <stdlib.h>
#include <cstdio>
#include <iostream>
#include <iomanip>
#include <cstring>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//> MAGMA
#include "magma_v2.h"
#include "magma_lapack.h"
#include "magma_internal.h"
#undef max
#undef min
#include "magma_templates.h"
#include "sync.cuh"
#undef max
#undef min
#include "shuffle.cuh"
#undef max
#undef min
#include "batched_kernel_param.h"

#include "magmaHC-kernels.hpp"

#include "../definitions.hpp"
#include "../gpu-idx-evals/dev-eval-indxing-5pt_rel_pos_geo_form_quat.cuh"
#include "../dev-cgesv-batched-small.cuh"
#include "../dev-get-new-data.cuh"

//namespace GPU_Device {

  template< unsigned Full_Parallel_Offset, \
            unsigned Partial_Parallel_Thread_Offset, \
            unsigned Partial_Parallel_Index_Offset, \
            unsigned Max_Order_of_t_Plus_One, \
            unsigned Partial_Parallel_Index_Offset_Hx, \
            unsigned Partial_Parallel_Index_Offset_Ht >
  __global__ void
  homotopy_continuation_solver_5pt_rel_pos_geo_form_quat(
    magmaFloatComplex** d_startSols_array, magmaFloatComplex** d_Track_array,
    magma_int_t* d_Hx_indices, magma_int_t* d_Ht_indices,
    magmaFloatComplex_ptr d_phc_coeffs_Hx, magmaFloatComplex_ptr d_phc_coeffs_Ht,
    bool* d_is_GPU_HC_Sol_Converge, bool* d_is_GPU_HC_Sol_Infinity,
    magmaFloatComplex* d_Debug_Purpose
  )
  {
    extern __shared__ magmaFloatComplex zdata[];
    const int tx = threadIdx.x;
    const int batchid = blockIdx.x ;

    magmaFloatComplex* d_startSols = d_startSols_array[batchid];
    magmaFloatComplex* d_track     = d_Track_array[batchid];
    const magmaFloatComplex* __restrict__ d_const_phc_coeffs_Hx = d_phc_coeffs_Hx;
    const magmaFloatComplex* __restrict__ d_const_phc_coeffs_Ht = d_phc_coeffs_Ht;
    const int* __restrict__ d_Hx_idx = d_Hx_indices;
    const int* __restrict__ d_Ht_idx = d_Ht_indices;

    //> Registers
    magmaFloatComplex r_cgesvA[NUM_OF_VARS] = {MAGMA_C_ZERO};
    magmaFloatComplex r_cgesvB = MAGMA_C_ZERO;
    int linfo = 0, rowid = tx;
    float t0 = 0.0, t_step = 0.0, delta_t = 0.05;
    bool inf_failed = 0;
    bool end_zone = 0;

    //> Shared Memory
    magmaFloatComplex *s_sols               = (magmaFloatComplex*)(zdata);
    magmaFloatComplex *s_track              = s_sols + (NUM_OF_VARS+1);
    magmaFloatComplex *s_track_last_success = s_track + (NUM_OF_VARS+1);
    magmaFloatComplex *sB                   = s_track_last_success + (NUM_OF_VARS+1);
    magmaFloatComplex *sx                   = sB + NUM_OF_VARS;
    magmaFloatComplex *s_phc_coeffs_Hx      = sx + NUM_OF_VARS;
    magmaFloatComplex *s_phc_coeffs_Ht      = s_phc_coeffs_Hx + (NUM_OF_COEFFS_FROM_PARAMS+1);
    float* dsx                              = (float*)(s_phc_coeffs_Ht + (NUM_OF_COEFFS_FROM_PARAMS+1));
    int* sipiv                              = (int*)(dsx + NUM_OF_VARS);
    float *s_sqrt_sols                      = (float*)(sipiv + NUM_OF_VARS);
    float *s_sqrt_corr                      = s_sqrt_sols + NUM_OF_VARS;
    float *s_norm                           = s_sqrt_corr + NUM_OF_VARS;
    bool s_isSuccessful                     = (bool)(s_norm + 2);
    int s_pred_success_count                = (int)(s_isSuccessful + 1);

    s_sols[tx] = d_startSols[tx];
    s_track[tx] = d_track[tx];
    s_track_last_success[tx] = s_track[tx];
    s_sqrt_sols[tx] = 0;
    s_sqrt_corr[tx] = 0;
    s_isSuccessful = 0;
    s_pred_success_count = 0;
    if (tx == 0) {
      s_sols[NUM_OF_VARS] = MAGMA_C_MAKE(1.0, 0.0);
      s_track[NUM_OF_VARS] = MAGMA_C_MAKE(1.0, 0.0);
      s_track_last_success[NUM_OF_VARS] = MAGMA_C_MAKE(1.0, 0.0);
    }
    __syncthreads();

    float one_half_delta_t;   // -- 1/2 \Delta t --

    #pragma unroll
    for (int step = 0; step <= HC_MAX_STEPS; step++) {
      if (t0 < 1.0 && (1.0-t0 > 0.0000001)) {

        // ===================================================================
        // -- Decide delta t at end zone --
        // ===================================================================
        if (!end_zone && fabs(1 - t0) <= (0.0500001)) {
          end_zone = true;
        }

        if (end_zone) {
          if (delta_t > fabs(1 - t0))
            delta_t = fabs(1 - t0);
        }
        else if (delta_t > fabs(1 - 0.05 - t0)) {
          delta_t = fabs(1 - 0.05 - t0);
        }

        t_step = t0;
        one_half_delta_t = 0.5 * delta_t;

        // ===================================================================
        // -- Runge-Kutta Predictor --
        // ===================================================================
        // -- get HxHt for k1 --
        eval_parameter_homotopy<Full_Parallel_Offset, Partial_Parallel_Thread_Offset, Partial_Parallel_Index_Offset, \
                                Max_Order_of_t_Plus_One, Partial_Parallel_Index_Offset_Hx, Partial_Parallel_Index_Offset_Ht> \
                                ( tx, t0, s_phc_coeffs_Hx, s_phc_coeffs_Ht, d_const_phc_coeffs_Hx, d_const_phc_coeffs_Ht );
        eval_Jacobian_Hx< HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS, NUM_OF_VARS*HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS>( tx, s_track, r_cgesvA, d_Hx_idx, s_phc_coeffs_Hx );
        eval_Jacobian_Ht< HT_MAXIMAL_TERMS*HT_MAXIMAL_PARTS >( tx, s_track, r_cgesvB, d_Ht_idx, s_phc_coeffs_Ht );

        // -- solve k1 --
        cgesv_batched_small_device< NUM_OF_VARS >( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
        magmablas_syncwarp();

        // -- compute x for the creation of HxHt for k2 --
        create_x_for_k2( tx, t0, delta_t, one_half_delta_t, s_sols, s_track, sB );
        magmablas_syncwarp();

        // -- get HxHt for k2 --
        eval_parameter_homotopy<Full_Parallel_Offset, Partial_Parallel_Thread_Offset, Partial_Parallel_Index_Offset, \
                                Max_Order_of_t_Plus_One, Partial_Parallel_Index_Offset_Hx, Partial_Parallel_Index_Offset_Ht> \
                                ( tx, t0, s_phc_coeffs_Hx, s_phc_coeffs_Ht, d_const_phc_coeffs_Hx, d_const_phc_coeffs_Ht );
        eval_Jacobian_Hx< HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS, NUM_OF_VARS*HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS>( tx, s_track, r_cgesvA, d_Hx_idx, s_phc_coeffs_Hx );
        eval_Jacobian_Ht< HT_MAXIMAL_TERMS*HT_MAXIMAL_PARTS >( tx, s_track, r_cgesvB, d_Ht_idx, s_phc_coeffs_Ht );

        // -- solve k2 --
        cgesv_batched_small_device< NUM_OF_VARS >( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
        magmablas_syncwarp();

        // -- compute x for the generation of HxHt for k3 --
        create_x_for_k3( tx, delta_t, one_half_delta_t, s_sols, s_track, s_track_last_success, sB );
        magmablas_syncwarp();

        // -- get HxHt for k3 --
        eval_Jacobian_Hx< HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS, NUM_OF_VARS*HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS>( tx, s_track, r_cgesvA, d_Hx_idx, s_phc_coeffs_Hx );
        eval_Jacobian_Ht< HT_MAXIMAL_TERMS*HT_MAXIMAL_PARTS >( tx, s_track, r_cgesvB, d_Ht_idx, s_phc_coeffs_Ht );

        // -- solve k3 --
        cgesv_batched_small_device< NUM_OF_VARS >( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
        magmablas_syncwarp();

        // -- compute x for the generation of HxHt for k4 --
        create_x_for_k4( tx, t0, delta_t, one_half_delta_t, s_sols, s_track, s_track_last_success, sB );
        magmablas_syncwarp();

        // -- get HxHt for k4 --
        eval_parameter_homotopy<Full_Parallel_Offset, Partial_Parallel_Thread_Offset, Partial_Parallel_Index_Offset, \
                                Max_Order_of_t_Plus_One, Partial_Parallel_Index_Offset_Hx, Partial_Parallel_Index_Offset_Ht> \
                                ( tx, t0, s_phc_coeffs_Hx, s_phc_coeffs_Ht, d_const_phc_coeffs_Hx, d_const_phc_coeffs_Ht );
        eval_Jacobian_Hx< HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS, NUM_OF_VARS*HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS>( tx, s_track, r_cgesvA, d_Hx_idx, s_phc_coeffs_Hx );
        eval_Jacobian_Ht< HT_MAXIMAL_TERMS*HT_MAXIMAL_PARTS >( tx, s_track, r_cgesvB, d_Ht_idx, s_phc_coeffs_Ht );

        // -- solve k4 --
        cgesv_batched_small_device< NUM_OF_VARS >( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
        magmablas_syncwarp();

        // -- make prediction --
        s_sols[tx] += sB[tx] * delta_t * 1.0/6.0;
        s_track[tx] = s_sols[tx];
        __syncthreads();

        // ===================================================================
        // -- Gauss-Newton Corrector --
        // ===================================================================
        //#pragma unroll
        for(int i = 0; i < HC_MAX_CORRECTION_STEPS; i++) {

          eval_Jacobian_Hx< HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS, NUM_OF_VARS*HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS>( tx, s_track, r_cgesvA, d_Hx_idx, s_phc_coeffs_Hx );
          eval_Homotopy< HT_MAXIMAL_TERMS*HT_MAXIMAL_PARTS >( tx, s_track, r_cgesvB, d_Ht_idx, s_phc_coeffs_Hx );

          // -- G-NUM_OF_VARS corrector first solve --
          cgesv_batched_small_device< NUM_OF_VARS >( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
          magmablas_syncwarp();

          // -- correct the sols --
          s_track[tx] -= sB[tx];
          __syncthreads();

          // -- compute the norms; norm[0] is norm(sB), norm[1] is norm(sol) --
          compute_norm2( tx, sB, s_track, s_sqrt_sols, s_sqrt_corr, s_norm );
          __syncthreads();

          s_isSuccessful = s_norm[0] < 0.000001 * s_norm[1];
          __syncthreads();

          if (s_isSuccessful)
	           break;
        }

        //> stop if the values of the solution is too large
        if ((s_norm[1] > 1e14) && (t0 < 1.0) && (1.0-t0 > 0.001)) {
          inf_failed = 1;
          break;
        }

        // ===================================================================
        // -- Decide Track Changes --
        // ===================================================================
        if (!s_isSuccessful) {
          s_pred_success_count = 0;
          delta_t *= 0.5;
          // -- should be the last successful tracked sols --
          s_track[tx] = s_track_last_success[tx];
          s_sols[tx] = s_track_last_success[tx];
          __syncthreads();
          t0 = t_step;
        }
        else {
          s_track_last_success[tx] = s_track[tx];
          s_sols[tx] = s_track[tx];
          __syncthreads();
          s_pred_success_count++;
          if (s_pred_success_count >= HC_NUM_OF_STEPS_TO_INCREASE_DELTA_T) {
            s_pred_success_count = 0;
            delta_t *= 2;
          }
        }
      }
      else {
        break;
      }
    }

    d_track[tx] = s_track[tx];
    if (tx == 0) {
      d_is_GPU_HC_Sol_Converge[ batchid ] = (t0 >= 1.0 || (1.0-t0 <= 0.0000001)) ? (1) : (0);
      d_is_GPU_HC_Sol_Infinity[ batchid ] = (inf_failed) ? (1) : (0);
    }

#if GPU_DEBUG
    d_Debug_Purpose[ batchid ] = (t0 >= 1.0 || (1.0-t0 <= 0.0000001)) ? MAGMA_C_MAKE(1.0, 0.0) : MAGMA_C_MAKE(t0, delta_t);
#endif
  }

  real_Double_t
  kernel_HC_Solver_5pt_rel_pos_geo_form_quat(
    magma_queue_t my_queue, \
    magmaFloatComplex** d_startSols_array, magmaFloatComplex** d_Track_array, \
    magma_int_t* d_Hx_idx_array,           magma_int_t* d_Ht_idx_array, \
    magmaFloatComplex_ptr d_phc_coeffs_Hx, magmaFloatComplex_ptr d_phc_coeffs_Ht, \
    bool* d_is_GPU_HC_Sol_Converge,        bool* d_is_GPU_HC_Sol_Infinity, \
    magmaFloatComplex* d_Debug_Purpose
  )
  {
    real_Double_t gpu_time;
    dim3 threads(NUM_OF_VARS, 1, 1);
    dim3 grid(NUM_OF_TRACKS, 1, 1);
    hipError_t e = hipErrorInvalidValue;

    //> Constant values for evaluating the Jacobians, passed as template
    const unsigned Full_Parallel_Offset                 = (NUM_OF_COEFFS_FROM_PARAMS+1)/(NUM_OF_VARS);
    const unsigned Partial_Parallel_Thread_Offset       = (NUM_OF_COEFFS_FROM_PARAMS+1) - (NUM_OF_VARS)*(Full_Parallel_Offset);
    const unsigned Partial_Parallel_Index_Offset        = (NUM_OF_VARS)*(Full_Parallel_Offset);
    const unsigned Max_Order_of_t_Plus_One              = MAX_ORDER_OF_T + 1;
    const unsigned Partial_Parallel_Index_Offset_for_Hx = (NUM_OF_VARS-1)*(Max_Order_of_t_Plus_One) + (MAX_ORDER_OF_T) + (Full_Parallel_Offset-1)*(Max_Order_of_t_Plus_One)*(NUM_OF_VARS) + 1;
    const unsigned Partial_Parallel_Index_Offset_for_Ht = (NUM_OF_VARS-1)*(MAX_ORDER_OF_T) + (MAX_ORDER_OF_T-1) + (Full_Parallel_Offset-1)*(MAX_ORDER_OF_T)*(NUM_OF_VARS) + 1;

    magma_int_t shmem  = 0;
    shmem += (NUM_OF_VARS+1) * sizeof(magmaFloatComplex);       // startSols
    shmem += (NUM_OF_VARS+1) * sizeof(magmaFloatComplex);       // track
    shmem += (NUM_OF_VARS+1) * sizeof(magmaFloatComplex);       // track_pred_init

    shmem += (NUM_OF_COEFFS_FROM_PARAMS+1) * sizeof(magmaFloatComplex);  // -- s_phc_coeffs_Hx --
    shmem += (NUM_OF_COEFFS_FROM_PARAMS+1) * sizeof(magmaFloatComplex);  // -- s_phc_coeffs_Ht --

    shmem += NUM_OF_VARS * sizeof(magmaFloatComplex); // sB
    shmem += NUM_OF_VARS * sizeof(magmaFloatComplex); // sx
    shmem += NUM_OF_VARS * sizeof(float);            // dsx
    shmem += NUM_OF_VARS * sizeof(int);               // pivot
    shmem += NUM_OF_VARS * sizeof(float);             // s_sqrt for sol norm-2 in G-NUM_OF_VARS corrector
    shmem += NUM_OF_VARS * sizeof(float);             // s_sqrt for corr norm-2 in G-NUM_OF_VARS corrector
    shmem += 2 * sizeof(float);             // s_norm for norm-2 in G-NUM_OF_VARS corrector
    shmem += 1 * sizeof(bool);              // is_successful 
    shmem += 1 * sizeof(int);               // predictor_success counter

    void *kernel_args[] = { &d_startSols_array, &d_Track_array, \
                            &d_Hx_idx_array, &d_Ht_idx_array, \
                            &d_phc_coeffs_Hx, &d_phc_coeffs_Ht, \
                            &d_is_GPU_HC_Sol_Converge, &d_is_GPU_HC_Sol_Infinity, \
                            &d_Debug_Purpose };

    gpu_time = magma_sync_wtime( my_queue );
    
    e = hipLaunchKernel((void*)homotopy_continuation_solver_5pt_rel_pos_geo_form_quat \
                          <Full_Parallel_Offset, \
                           Partial_Parallel_Thread_Offset, \
                           Partial_Parallel_Index_Offset, \
                           Max_Order_of_t_Plus_One, \
                           Partial_Parallel_Index_Offset_for_Hx, \
                           Partial_Parallel_Index_Offset_for_Ht>, \
                          grid, threads, kernel_args, shmem, my_queue->hip_stream());

    gpu_time = magma_sync_wtime( my_queue ) - gpu_time;
    if( e != hipSuccess ) printf("hipLaunchKernel of homotopy_continuation_solver_5pt_rel_pos_geo_form_quat is not successful!\n");

    return gpu_time;
  }

//}

#endif
