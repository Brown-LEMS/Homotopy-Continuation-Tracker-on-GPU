#include "hip/hip_runtime.h"
#ifndef kernel_HC_Solver_5pt_rel_pos_alg_form_quat_cu
#define kernel_HC_Solver_5pt_rel_pos_alg_form_quat_cu
// =======================================================================================
// GPU homotopy continuation solver for 5-point relative pose problem (Algebraic Form)
//
// Modifications
//    Chiang-Heng Chien  22-11-16:   Initially created
//    Chiang-Heng Chien  24-01-04:   Add macro definitions for computing coefficients from parameter homotopy
//
//> (c) LEMS, Brown University
//> Chiang-Heng Chien (chiang-heng_chien@brown.edu)
// =======================================================================================
#include <stdio.h>
#include <stdlib.h>
#include <cstdio>
#include <iostream>
#include <iomanip>
#include <cstring>

// cuda included
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// magma
#include "flops.h"
#include "magma_v2.h"
#include "magma_lapack.h"
#include "magma_internal.h"
#undef max
#undef min
#include "magma_templates.h"
#include "sync.cuh"
#undef max
#undef min
#include "shuffle.cuh"
#undef max
#undef min
#include "batched_kernel_param.h"

#include "../definitions.hpp"
#include "magmaHC-kernels.hpp"

//> device function
#include "../gpu-idx-evals/dev-eval-indxing-5pt_rel_pos_alg_form_quat.cuh"
#include "../dev-cgesv-batched-small.cuh"
#include "../dev-get-new-data.cuh"

template< unsigned Full_Parallel_Offset, \
          unsigned Partial_Parallel_Thread_Offset, \
          unsigned Partial_Parallel_Index_Offset, \
          unsigned Max_Order_of_t_Plus_One, \
          unsigned Partial_Parallel_Index_Offset_Hx, \
          unsigned Partial_Parallel_Index_Offset_Ht >
__global__ void
homotopy_continuation_solver_5pt_rel_pos_alg_form_quat(
  magmaFloatComplex** d_startSols_array, magmaFloatComplex** d_Track_array,
  magma_int_t* d_Hx_indices, magma_int_t* d_Ht_indices,
  magmaFloatComplex_ptr d_phc_coeffs_Hx, magmaFloatComplex_ptr d_phc_coeffs_Ht,
  bool* d_is_GPU_HC_Sol_Converge, bool* d_is_GPU_HC_Sol_Infinity,
  magmaFloatComplex* d_Debug_Purpose
)
{
  extern __shared__ magmaFloatComplex zdata[];
  const int tx = threadIdx.x;
  const int batchid = blockIdx.x ;

  magmaFloatComplex* d_startSols   = d_startSols_array[batchid];
  magmaFloatComplex* d_track       = d_Track_array[batchid];
  const int* __restrict__ d_Hx_idx = d_Hx_indices;
  const int* __restrict__ d_Ht_idx = d_Ht_indices;
  const magmaFloatComplex* __restrict__ d_const_phc_coeffs_Hx = d_phc_coeffs_Hx;
  const magmaFloatComplex* __restrict__ d_const_phc_coeffs_Ht = d_phc_coeffs_Ht;
  
  //> registers declarations
  magmaFloatComplex r_cgesvA[NUM_OF_VARS] = {MAGMA_C_ZERO};
  magmaFloatComplex r_cgesvB = MAGMA_C_ZERO;
  int linfo = 0, rowid = tx;
  float t0 = 0.0, t_step = 0.0, delta_t = 0.05;
  bool end_zone = 0;

  //> shared memory declarations
  magmaFloatComplex *s_sols               = (magmaFloatComplex*)(zdata);
  magmaFloatComplex *s_track              = s_sols + (NUM_OF_VARS+1);
  magmaFloatComplex *s_track_last_success = s_track + (NUM_OF_VARS+1);
  magmaFloatComplex *sB                   = s_track_last_success + (NUM_OF_VARS+1);
  magmaFloatComplex *sx                   = sB + NUM_OF_VARS;
  magmaFloatComplex *s_phc_coeffs_Hx      = sx + NUM_OF_VARS;
  magmaFloatComplex *s_phc_coeffs_Ht      = s_phc_coeffs_Hx + (NUM_OF_COEFFS_FROM_PARAMS+1);
  float* dsx                              = (float*)(s_phc_coeffs_Ht + (NUM_OF_COEFFS_FROM_PARAMS+1));
  int* sipiv                              = (int*)(dsx + NUM_OF_VARS);
  int s_pred_success_count                = (int)(sipiv + NUM_OF_VARS);

  s_sols[tx] = d_startSols[tx];
  s_track[tx] = d_track[tx];
  s_track_last_success[tx] = s_track[tx];
  s_pred_success_count = 0;
  if (tx == 0) {
    s_sols[NUM_OF_VARS]               = MAGMA_C_MAKE(1.0, 0.0);
    s_track[NUM_OF_VARS]              = MAGMA_C_MAKE(1.0, 0.0);
    s_track_last_success[NUM_OF_VARS] = MAGMA_C_MAKE(1.0, 0.0);
  }
  __syncthreads();

  float one_half_delta_t;   //> 1/2 \Delta t
  float r_sqrt_sols;
  float r_sqrt_corr;
  bool r_isSuccessful;
  bool r_isInfFail;
#if APPLY_GAMMA_TRICK
  magmaFloatComplex gammified_t0;
  magmaFloatComplex gammified_t0_plus_dt;
  magmaFloatComplex gammified_t0_plus_one_half_dt;
#endif

  #pragma unroll
  for (int step = 0; step <= HC_MAX_STEPS; step++) {
    if (t0 < 1.0 && (1.0-t0 > 0.0000001)) {

      // ===================================================================
      //> Decide delta t at end zone
      // ===================================================================
      if (!end_zone && fabs(1 - t0) <= (0.0500001)) {
        end_zone = true;
      }

      if (end_zone) {
        if (delta_t > fabs(1 - t0))
          delta_t = fabs(1 - t0);
      }
      else if (delta_t > fabs(1 - 0.05 - t0)) {
        delta_t = fabs(1 - 0.05 - t0);
      }

      t_step = t0;
      one_half_delta_t = 0.5 * delta_t;
      // ===================================================================
      //> Runge-Kutta Predictor
      // ===================================================================
#if APPLY_GAMMA_TRICK
      gammified_t0                  = GAMMA * t0 / (1.0 + (GAMMA - 1.0) * t0);                                      //> t0
      gammified_t0_plus_dt          = GAMMA * (t0 + delta_t) / (1.0 + (GAMMA - 1.0) * (t0 + delta_t));              //> t1
      gammified_t0_plus_one_half_dt = GAMMA * (t0 + one_half_delta_t) / (1.0 + (GAMMA - 1.0) * (t0 + one_half_delta_t));  //> t05
#endif
        //> get HxHt for k1
#if APPLY_GAMMA_TRICK
      eval_parameter_homotopy<magmaFloatComplex, Full_Parallel_Offset, Partial_Parallel_Thread_Offset, Partial_Parallel_Index_Offset, \
                              Max_Order_of_t_Plus_One, Partial_Parallel_Index_Offset_Hx, Partial_Parallel_Index_Offset_Ht> \
                              ( tx, gammified_t0, s_phc_coeffs_Hx, s_phc_coeffs_Ht, d_const_phc_coeffs_Hx, d_const_phc_coeffs_Ht );
#else
      eval_parameter_homotopy<float, Full_Parallel_Offset, Partial_Parallel_Thread_Offset, Partial_Parallel_Index_Offset, \
                              Max_Order_of_t_Plus_One, Partial_Parallel_Index_Offset_Hx, Partial_Parallel_Index_Offset_Ht> \
                              ( tx, t0, s_phc_coeffs_Hx, s_phc_coeffs_Ht, d_const_phc_coeffs_Hx, d_const_phc_coeffs_Ht );
#endif
      eval_Jacobian_Hx< HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS, NUM_OF_VARS*HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS>( tx, s_track, r_cgesvA, d_Hx_idx, s_phc_coeffs_Hx );
      eval_Jacobian_Ht< HT_MAXIMAL_TERMS*HT_MAXIMAL_PARTS >( tx, s_track, r_cgesvB, d_Ht_idx, s_phc_coeffs_Ht );

      //> solve k1
      cgesv_batched_small_device< NUM_OF_VARS >( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
      magmablas_syncwarp();

        //> compute x for the creation of HxHt for k2 and get HxHt for k2
#if APPLY_GAMMA_TRICK
      magmaFloatComplex gc = GAMMA / (((GAMMA - 1.0) * t0 + 1.0) * ((GAMMA - 1.0) * t0 + 1.0));
      create_x_for_k2( tx, t0, delta_t, one_half_delta_t, s_sols, s_track, sB, gc );
      magmablas_syncwarp();
      eval_parameter_homotopy<magmaFloatComplex, Full_Parallel_Offset, Partial_Parallel_Thread_Offset, Partial_Parallel_Index_Offset, \
                              Max_Order_of_t_Plus_One, Partial_Parallel_Index_Offset_Hx, Partial_Parallel_Index_Offset_Ht> \
                              ( tx, gammified_t0_plus_one_half_dt, s_phc_coeffs_Hx, s_phc_coeffs_Ht, d_const_phc_coeffs_Hx, d_const_phc_coeffs_Ht );
#else
      create_x_for_k2( tx, t0, delta_t, one_half_delta_t, s_sols, s_track, sB, MAGMA_C_ONE );
      magmablas_syncwarp();
      eval_parameter_homotopy<float, Full_Parallel_Offset, Partial_Parallel_Thread_Offset, Partial_Parallel_Index_Offset, \
                              Max_Order_of_t_Plus_One, Partial_Parallel_Index_Offset_Hx, Partial_Parallel_Index_Offset_Ht> \
                              ( tx, t0, s_phc_coeffs_Hx, s_phc_coeffs_Ht, d_const_phc_coeffs_Hx, d_const_phc_coeffs_Ht );
#endif
      eval_Jacobian_Hx< HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS, NUM_OF_VARS*HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS>( tx, s_track, r_cgesvA, d_Hx_idx, s_phc_coeffs_Hx );
      eval_Jacobian_Ht< HT_MAXIMAL_TERMS*HT_MAXIMAL_PARTS >( tx, s_track, r_cgesvB, d_Ht_idx, s_phc_coeffs_Ht );

      //> solve k2
      cgesv_batched_small_device< NUM_OF_VARS >( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
      magmablas_syncwarp();

#if APPLY_GAMMA_TRICK
      magmaFloatComplex gc05 = GAMMA / (((GAMMA - 1.0) * (t0 + one_half_delta_t) + 1.0) * ((GAMMA - 1.0) * (t0 + one_half_delta_t) + 1.0));
      create_x_for_k3( tx, delta_t, one_half_delta_t, s_sols, s_track, s_track_last_success, sB, gc05 );
      magmablas_syncwarp();
#else
      create_x_for_k3( tx, delta_t, one_half_delta_t, s_sols, s_track, s_track_last_success, sB, MAGMA_C_ONE );
      magmablas_syncwarp();
#endif
      //> get HxHt for k3
      eval_Jacobian_Hx< HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS, NUM_OF_VARS*HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS>( tx, s_track, r_cgesvA, d_Hx_idx, s_phc_coeffs_Hx );
      eval_Jacobian_Ht< HT_MAXIMAL_TERMS*HT_MAXIMAL_PARTS >( tx, s_track, r_cgesvB, d_Ht_idx, s_phc_coeffs_Ht );

      //> solve k3
      cgesv_batched_small_device< NUM_OF_VARS >( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
      magmablas_syncwarp();

      //> compute x for the generation of HxHt for k4
#if APPLY_GAMMA_TRICK
      create_x_for_k4( tx, t0, delta_t, one_half_delta_t, s_sols, s_track, s_track_last_success, sB, gc05 );
      magmablas_syncwarp();
      //> get HxHt for k4
      eval_parameter_homotopy<magmaFloatComplex, Full_Parallel_Offset, Partial_Parallel_Thread_Offset, Partial_Parallel_Index_Offset, \
                              Max_Order_of_t_Plus_One, Partial_Parallel_Index_Offset_Hx, Partial_Parallel_Index_Offset_Ht> \
                              ( tx, gammified_t0_plus_dt, s_phc_coeffs_Hx, s_phc_coeffs_Ht, d_const_phc_coeffs_Hx, d_const_phc_coeffs_Ht );
#else
      create_x_for_k4( tx, t0, delta_t, one_half_delta_t, s_sols, s_track, s_track_last_success, sB, MAGMA_C_ONE );
      magmablas_syncwarp();
      //> get HxHt for k4
      eval_parameter_homotopy<float, Full_Parallel_Offset, Partial_Parallel_Thread_Offset, Partial_Parallel_Index_Offset, \
                              Max_Order_of_t_Plus_One, Partial_Parallel_Index_Offset_Hx, Partial_Parallel_Index_Offset_Ht> \
                              ( tx, t0, s_phc_coeffs_Hx, s_phc_coeffs_Ht, d_const_phc_coeffs_Hx, d_const_phc_coeffs_Ht );
#endif
      eval_Jacobian_Hx< HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS, NUM_OF_VARS*HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS>( tx, s_track, r_cgesvA, d_Hx_idx, s_phc_coeffs_Hx );
      eval_Jacobian_Ht< HT_MAXIMAL_TERMS*HT_MAXIMAL_PARTS >( tx, s_track, r_cgesvB, d_Ht_idx, s_phc_coeffs_Ht );

      //> solve k4
      cgesv_batched_small_device< NUM_OF_VARS >( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
      magmablas_syncwarp();

      //> make prediction
#if APPLY_GAMMA_TRICK
      magmaFloatComplex gc1 = GAMMA / (((GAMMA - 1.0) * (t0 + delta_t) + 1.0) * ((GAMMA - 1.0) * (t0 + delta_t) + 1.0));
      s_sols[tx] += sB[tx] * delta_t * gc1 * 1.0/6.0;
      s_track[tx] = s_sols[tx];
      __syncthreads();
#else
      s_sols[tx] += sB[tx] * delta_t * 1.0/6.0;
      s_track[tx] = s_sols[tx];
      __syncthreads();
#endif

      // ===================================================================
      //> Gauss-Newton Corrector
      // ===================================================================
      //#pragma unroll
      for(int i = 0; i < HC_MAX_CORRECTION_STEPS; i++) {

        eval_Jacobian_Hx< HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS, NUM_OF_VARS*HX_MAXIMAL_TERMS*HX_MAXIMAL_PARTS >( tx, s_track, r_cgesvA, d_Hx_idx, s_phc_coeffs_Hx );
        eval_Homotopy< HT_MAXIMAL_TERMS*HT_MAXIMAL_PARTS >( tx, s_track, r_cgesvB, d_Ht_idx, s_phc_coeffs_Hx );

        //> G-N corrector first solve
        cgesv_batched_small_device< NUM_OF_VARS >( tx, r_cgesvA, sipiv, r_cgesvB, sB, sx, dsx, rowid, linfo );
        magmablas_syncwarp();

        //> correct the sols
        s_track[tx] -= sB[tx];
        __syncthreads();

        r_sqrt_sols = MAGMA_C_REAL(sB[tx])*MAGMA_C_REAL(sB[tx]) + MAGMA_C_IMAG(sB[tx])*MAGMA_C_IMAG(sB[tx]);
        r_sqrt_corr = MAGMA_C_REAL(s_track[tx])*MAGMA_C_REAL(s_track[tx]) + MAGMA_C_IMAG(s_track[tx])*MAGMA_C_IMAG(s_track[tx]);
        __syncthreads();

        for (int offset = WARP_SIZE/2; offset > 0; offset /= 2 ) {
            r_sqrt_sols += __shfl_down_sync(__activemask(), r_sqrt_sols, offset);
            r_sqrt_corr += __shfl_down_sync(__activemask(), r_sqrt_corr, offset);
        }

        if ( tx == 0 ) {
            r_isSuccessful = r_sqrt_sols < 0.000001 * r_sqrt_corr;
            r_isInfFail = (r_sqrt_corr > 1e14) ? (true) : (false);
        }
        //> Broadcast the values of r_isSuccessful and r_isInfFail from thread 0 to all the rest of the threads
        r_isSuccessful = __shfl_sync(__activemask(), r_isSuccessful, 0);
        r_isInfFail = __shfl_sync(__activemask(), r_isInfFail, 0);

        if (r_isInfFail) break;
        if (r_isSuccessful) break;
      }

      if ( r_isInfFail ) break;

      // ===================================================================
      //> Decide Track Changes
      // ===================================================================
      if (!r_isSuccessful) {
        s_pred_success_count = 0;
        delta_t *= 0.5;
        //> should be the last successful tracked sols
        s_track[tx] = s_track_last_success[tx];
        s_sols[tx] = s_track_last_success[tx];
        __syncthreads();
        t0 = t_step;
      }
      else {
        s_track_last_success[tx] = s_track[tx];
        s_sols[tx] = s_track[tx];
        __syncthreads();
        s_pred_success_count++;
        if (s_pred_success_count >= HC_NUM_OF_STEPS_TO_INCREASE_DELTA_T) {
          s_pred_success_count = 0;
          delta_t *= 2;
        }
      }
    }
    else {
      break;
    }
  }
  
  d_track[tx] = s_track[tx];
  if (tx == 0) {
    d_is_GPU_HC_Sol_Converge[ batchid ] = (t0 >= 1.0 || (1.0-t0 <= 0.0000001)) ? (1) : (0);
    d_is_GPU_HC_Sol_Infinity[ batchid ] = (r_isInfFail) ? (1) : (0);
  }

#if GPU_DEBUG
  d_Debug_Purpose[ batchid ] = (t0 >= 1.0 || (1.0-t0 <= 0.0000001)) ? MAGMA_C_MAKE(1.0, 0.0) : MAGMA_C_MAKE(t0, delta_t);
#endif
}

real_Double_t
kernel_HC_Solver_5pt_rel_pos_alg_form_quat(                      
  magma_queue_t my_queue, \
  magmaFloatComplex** d_startSols_array, magmaFloatComplex** d_Track_array, \
  magma_int_t* d_Hx_idx_array,           magma_int_t* d_Ht_idx_array, \
  magmaFloatComplex_ptr d_phc_coeffs_Hx, magmaFloatComplex_ptr d_phc_coeffs_Ht, \
  bool* d_is_GPU_HC_Sol_Converge,        bool* d_is_GPU_HC_Sol_Infinity, \
  magmaFloatComplex* d_Debug_Purpose
)
{
  real_Double_t gpu_time;
  dim3 threads(NUM_OF_VARS, 1, 1);
  dim3 grid(NUM_OF_TRACKS, 1, 1);
  hipError_t e = hipErrorInvalidValue;

  //> Constant values for evaluating the Jacobians, passed as template
  const unsigned Full_Parallel_Offset                 = (NUM_OF_COEFFS_FROM_PARAMS+1)/(NUM_OF_VARS);
  const unsigned Partial_Parallel_Thread_Offset       = (NUM_OF_COEFFS_FROM_PARAMS+1) - (NUM_OF_VARS)*(Full_Parallel_Offset);
  const unsigned Partial_Parallel_Index_Offset        = (NUM_OF_VARS)*(Full_Parallel_Offset);
  const unsigned Max_Order_of_t_Plus_One              = MAX_ORDER_OF_T + 1;
  const unsigned Partial_Parallel_Index_Offset_for_Hx = (NUM_OF_VARS-1)*(Max_Order_of_t_Plus_One) + (MAX_ORDER_OF_T) + (Full_Parallel_Offset-1)*(Max_Order_of_t_Plus_One)*(NUM_OF_VARS) + 1;
  const unsigned Partial_Parallel_Index_Offset_for_Ht = (NUM_OF_VARS-1)*(MAX_ORDER_OF_T) + (MAX_ORDER_OF_T-1) + (Full_Parallel_Offset-1)*(MAX_ORDER_OF_T)*(NUM_OF_VARS) + 1;

  //> declare shared memory
  magma_int_t shmem  = 0;
  shmem += (NUM_OF_VARS+1) * sizeof(magmaFloatComplex);                 // startSols
  shmem += (NUM_OF_VARS+1) * sizeof(magmaFloatComplex);                 // track
  shmem += (NUM_OF_VARS+1) * sizeof(magmaFloatComplex);                 // track_pred_init
  shmem += (NUM_OF_COEFFS_FROM_PARAMS+1) * sizeof(magmaFloatComplex);   //> s_phc_coeffs_Hx
  shmem += (NUM_OF_COEFFS_FROM_PARAMS+1) * sizeof(magmaFloatComplex);   //> s_phc_coeffs_Ht
  shmem += NUM_OF_VARS * sizeof(magmaFloatComplex);                     // sB
  shmem += NUM_OF_VARS * sizeof(magmaFloatComplex);                     // sx
  shmem += NUM_OF_VARS * sizeof(float);                                 // dsx
  shmem += NUM_OF_VARS * sizeof(int);                                   // pivot
  shmem += 1 * sizeof(int);                                             // predictor_success counter

  void *kernel_args[] = { &d_startSols_array, &d_Track_array, \
                          &d_Hx_idx_array, &d_Ht_idx_array, \
                          &d_phc_coeffs_Hx, &d_phc_coeffs_Ht, \
                          &d_is_GPU_HC_Sol_Converge, &d_is_GPU_HC_Sol_Infinity, \
                          &d_Debug_Purpose };

  gpu_time = magma_sync_wtime( my_queue );

  // float gpu_time_cost;
  // hipEvent_t start, stop;
  // cudacheck( hipEventCreate(&start) );
  // cudacheck( hipEventCreate(&stop) );

  // cudacheck( hipEventRecord(start) );
  e = hipLaunchKernel((void*)homotopy_continuation_solver_5pt_rel_pos_alg_form_quat \
                        <Full_Parallel_Offset, \
                          Partial_Parallel_Thread_Offset, \
                          Partial_Parallel_Index_Offset, \
                          Max_Order_of_t_Plus_One, \
                          Partial_Parallel_Index_Offset_for_Hx, \
                          Partial_Parallel_Index_Offset_for_Ht>, \
                        grid, threads, kernel_args, shmem, my_queue->hip_stream());

  // cudacheck( hipEventRecord(stop) );
  // cudacheck( hipEventSynchronize(stop) );
  // cudacheck( hipEventElapsedTime(&gpu_time_cost, start, stop) );

  gpu_time = magma_sync_wtime( my_queue ) - gpu_time;
  if( e != hipSuccess ) printf("hipLaunchKernel of homotopy_continuation_solver_5pt_rel_pos_alg_form_quat is not successful!\n");

  return gpu_time;
  // return gpu_time_cost;
}

#endif
